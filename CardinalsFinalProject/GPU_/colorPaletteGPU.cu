#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <complex>
#include <cmath>
#include <sys/time.h>

#define NUM_COLORS 143
#define THREADS_PER_BLOCK 960

using namespace cv;
using namespace std;

__const__ char* COLORNAMES[NUM_COLORS] = {
	"INDIANRED", "LIGHTCORAL", "SALMON", "DARKSALMON", "LIGHTSALMON", "CRIMSON", "RED", "FIREBRICK", "DARKRED", "PINK",
"LIGHTPINK","HOTPINK","DEEPPINK","MEDIUMVIOLETRED","PALEVIOLETRED","LIGHTSALMON","CORAL","TOMATO","ORANGERED",
"DARKORANGE","ORANGE","GOLD","YELLOW","LIGHTYELLOW","LEMONCHIFFON","LIGHTGOLDENRODYELLOW","PAPAYAWHIP","MOCCASIN",
"PEACHPUFF","PALEGOLDENROD","KHAKI","DARKKHAKI","LAVENDER","THISTLE","PLUM","VIOLET","ORCHID","FUCHSIA","MAGENTA",
"MEDIUMORCHID","MEDIUMPURPLE","REBECCAPURPLE","BLUEVIOLET","DARKVIOLET","DARKORCHID","DARKMAGENTA","PURPLE","INDIGO",
"SLATEBLUE","DARKSLATEBLUE","MEDIUMSLATEBLUE","GREENYELLOW","CHARTREUSE","LAWNGREEN","LIME","LIMEGREEN","PALEGREEN",
"LIGHTGREEN","MEDIUMSPRINGGREEN","SPRINGGREEN","MEDIUMSEAGREEN","SEAGREEN","FORESTGREEN","GREEN","DARKGREEN","YELLOWGREEN",
"OLIVEDRAB","OLIVE","DARKOLIVEGREEN","MEDIUMAQUAMARINE","DARKSEAGREEN","LIGHTSEAGREEN","DARKCYAN","TEAL","AQUA","CYAN",
"LIGHTCYAN","PALETURQUOISE","AQUAMARINE","TURQUOISE","MEDIUMTURQUOISE","DARKTURQUOISE","CADETBLUE","STEELBLUE","LIGHTSTEELBLUE",
"POWDERBLUE","LIGHTBLUE","SKYBLUE","LIGHTSKYBLUE","DEEPSKYBLUE","DODGERBLUE","CORNFLOWERBLUE","MEDIUMSLATEBLUE","ROYALBLUE",
"BLUE","MEDIUMBLUE","DARKBLUE","NAVY","MIDNIGHTBLUE","CORNSILK","BLANCHEDALMOND","BISQUE","NAVAJOWHITE","WHEAT","BURLYWOOD",
"TAN","ROSYBROWN","SANDYBROWN","GOLDENROD","DARKGOLDENROD","PERU","CHOCOLATE","SADDLEBROWN","SIENNA","BROWN","MAROON","WHITE",
"SNOW","HONEYDEW","MINTCREAM","AZURE","ALICEBLUE","GHOSTWHITE","WHITESMOKE","SEASHELL","BEIGE","OLDLACE","FLORALWHITE","IVORY",
"ANTIQUEWHITE","LINEN","LAVENDERBLUSH","MISTYROSE","GAINSBORO","LIGHTGRAY","SILVER","DARKGRAY","GRAY","DIMGRAY","LIGHTSLATEGRAY",
"SLATEGRAY","DARKSLATEGRAY","BLACK" };

__constant__ uchar RED[NUM_COLORS] =
{ 205,240,250,233,255,220,255,178,139,255,255,255,255,199,219,255,255,255,255,255,255,255,255,255,255,250,255,255,255,238,240,189,230,
216,221,238,218,255,255,186,147,102,138,148,153,139,128,75,106,72,123,173,127,124,0,50,152,144,0,0,60,46,34,0,0,154,107,128,85,102,143,
32,0,0,0,0,224,175,127,64,72,0,95,70,176,176,173,135,135,0,30,100,123,65,0,0,0,0,25,255,255,255,255,245,222,210,188,244,218,184,205,210,
139,160,165,128,255,255,240,245,240,240,248,245,255,245,253,255,255,250,250,255,255,220,211,192,169,128,105,119,112,47,0 };

__constant__ uchar GREEN[NUM_COLORS] =
{ 92,128,128,150,160,20,0,34,0,192,182,105,20,21,112,160,127,99,69,140,165,215,255,255,250,250,239,228,218,232,230,183,230,191,160,130,112,
0,0,85,112,51,43,0,50,0,0,0,90,61,104,255,255,252,255,205,251,238,250,255,179,139,139,128,100,205,142,128,107,205,188,178,139,128,255,255,
255,238,255,224,209,206,158,130,196,224,216,206,206,191,144,149,104,105,0,0,0,0,25,248,235,228,222,222,184,180,143,164,165,134,133,105,69,
82,42,0,255,250,255,255,255,248,248,245,245,245,245,250,255,235,240,240,228,220,211,192,169,128,105,136,128,79,0 };

__constant__ uchar BLUE[NUM_COLORS] =
{ 92,128,114,122,122,60,0,34,0,203,193,180,147,133,147,122,80,71,0,0,0,0,0,224,205,210,213,181,185,170,140,107,250,216,221,238,214,255,255,211,
219,153,226,211,204,139,128,130,205,139,238,47,0,0,0,50,152,144,154,127,113,87,34,0,0,50,35,0,47,170,139,170,139,128,255,255,255,238,212,208,
204,209,160,180,222,230,230,235,250,255,255,237,238,225,255,205,139,128,112,220,205,196,173,179,135,140,143,96,32,11,63,30,19,45,42,0,255,250,
240,250,255,255,255,245,238,220,230,240,240,215,230,245,225,220,211,192,169,128,105,153,144,79,0 };




__global__ void frqColorArrayBuilder(uchar* gFREQINDEXES, uchar* gpixelBArr,uchar* gpixelGArr,uchar* gpixelRArr, int totalPixels) {	
	uint minIndex = 0;
	int freqIndexesSize = blockIdx.x*blockDim.x + threadIdx.x;;
	uchar avg;
	if(freqIndexesSize < totalPixels)
	{		
		//initialize minAvg to the first color
		uchar minAvg = (fabsf(gpixelBArr[freqIndexesSize] - BLUE[0]) + fabsf(gpixelGArr[freqIndexesSize]) + fabsf(gpixelRArr[freqIndexesSize])) / 3;
		
			for (int i = 1; i < NUM_COLORS; i++) //iterate through the array of 143 color codes
			{
				//calculate the avg for the passed pixel values and the current color code (i)
				avg = (fabsf(gpixelBArr[freqIndexesSize] - BLUE[i]) + fabsf(gpixelGArr[freqIndexesSize]- GREEN[i]) + fabsf(gpixelRArr[freqIndexesSize] - RED[i])) / 3;

				//find our lowest avg
				if (avg < minAvg) //if compareNum is less than lowest min (min0)
				{
					minIndex = i; //save the index of the color code from the COLORNAMES array
					minAvg = avg; //save the new minAvg for future comparisons
				}
			}
			gFREQINDEXES[freqIndexesSize] = minIndex; //populate the FREQINDEXES array
	}
}



struct timeval start, end;
void starttime() {
	gettimeofday(&start, 0);
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

int main( int argc, char** argv )
{	//get BGR values from image
    Mat image;
    image = imread("greens.jpg", CV_LOAD_IMAGE_COLOR);   // Read the file
    if(! image.data ) { // Check for invalid input
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }
	int totalPixels = image.total();
	
	uchar* pixelBArr = (uchar*) malloc(totalPixels * sizeof(uchar)); //array holding image's pixel B values
	uchar* pixelGArr = (uchar*) malloc(totalPixels * sizeof(uchar)); //array holding image's pixel G values
	uchar* pixelRArr = (uchar*) malloc(totalPixels * sizeof(uchar));; //array holding image's pixel R values
	int currentBSize = 0;
	int currentGSize = 0;
	int currentRSize = 0;
	int currentIndex = 0;
	int i;
    for (i = 0; i < 3*image.total(); i++)
	{
		if(currentIndex == 0) { //if a B value
			currentIndex++;
			pixelBArr[currentBSize] = (uchar)image.data[i]; //put the pixel's b value into b array
			currentBSize++;
		}
		else if(currentIndex == 1) { //if a G value
			currentIndex++;
			pixelGArr[currentGSize] = (uchar)image.data[i];//put the pixel's g value into g array
			currentGSize++;
		}
		else if(currentIndex == 2) { //if a R value
			currentIndex++;
			pixelRArr[currentRSize] = (uchar)image.data[i];//put the pixel's r value into r array
			currentRSize++;
			currentIndex = 0;
		}
		else{
			printf("currentIndex is not 0 1 or 2");
			currentIndex = 0;
		}
		
	}
	///////////////////////////////////////////////////////////////////
	starttime(); //START THE TIMER
	//Create array of totalPixels size that will hold each pixel's closest HTML color RGB value
	uchar* FREQINDEXES = (uchar*) malloc(totalPixels * sizeof(uint)); //array for each pixel's closest color index	
	uchar* gpu_pixelBArr = (uchar*) malloc(totalPixels * sizeof(uchar));
	uchar* gpu_pixelGArr = (uchar*) malloc(totalPixels * sizeof(uchar));
	uchar* gpu_pixelRArr = (uchar*) malloc(totalPixels * sizeof(uchar));
	
	//allocate GPU memory
	uchar* gpu_FREQINDEXES;
	hipMalloc(&gpu_FREQINDEXES, totalPixels*sizeof(uchar));
	hipMalloc(&gpu_pixelBArr, totalPixels*sizeof(uchar));
	hipMalloc(&gpu_pixelGArr, totalPixels*sizeof(uchar));
	hipMalloc(&gpu_pixelRArr, totalPixels*sizeof(uchar));
	
	
	//copy cpu to gpu
	hipMemcpy(gpu_FREQINDEXES , FREQINDEXES, totalPixels*sizeof( uchar ) , hipMemcpyHostToDevice );
	hipMemcpy(gpu_pixelBArr , pixelBArr, totalPixels*sizeof(uchar) , hipMemcpyHostToDevice );
	hipMemcpy(gpu_pixelGArr , pixelGArr, totalPixels*sizeof(uchar) , hipMemcpyHostToDevice );
	hipMemcpy(gpu_pixelRArr , pixelRArr, totalPixels*sizeof(uchar) , hipMemcpyHostToDevice );
	
	
	//gpu function call
	int numblocks = totalPixels / THREADS_PER_BLOCK;
	if(totalPixels % THREADS_PER_BLOCK != 0) {
		numblocks++;
	}
	frqColorArrayBuilder<<<numblocks , THREADS_PER_BLOCK>>>(gpu_FREQINDEXES, gpu_pixelBArr, gpu_pixelGArr, gpu_pixelRArr, totalPixels);
	
	
	//copy gpu to cpu
	hipMemcpy(FREQINDEXES , gpu_FREQINDEXES, totalPixels*sizeof( uchar ) , hipMemcpyDeviceToHost );
	hipMemcpy(pixelBArr , gpu_pixelBArr, totalPixels*sizeof(uchar) , hipMemcpyDeviceToHost );
	hipMemcpy(pixelGArr , gpu_pixelGArr, totalPixels*sizeof(uchar) , hipMemcpyDeviceToHost );
	hipMemcpy(pixelRArr , gpu_pixelRArr, totalPixels*sizeof(uchar) , hipMemcpyDeviceToHost );
	
	
	hipFree(gpu_FREQINDEXES);
	hipFree(gpu_pixelBArr);
	hipFree(gpu_pixelGArr);
	hipFree(gpu_pixelRArr);
	free(pixelBArr);
	free(pixelGArr);
	free(pixelRArr);
	
	///////////////////////////////////////////////////////////////
	int* COUNTER = (int*) malloc(NUM_COLORS * sizeof(int)); //array acting as a counter for how many times each of the HTML color indexes appears in FREQINDEXES
	//initialize all COUNTER values to 0
	for(int i = 0; i < NUM_COLORS; i++) {
		COUNTER[i] = 0; 
	}
	
	///////////////////////////////////////////////////////////////////////
	//populate COUNTER array to show how many times each color shows up in image
	for (int i = 0;  i < NUM_COLORS; i++) { //COLORNAMES (indexes) 
		for(int j = 0; j < totalPixels; j++) { //FREQINDEXES 
			if (i == FREQINDEXES[j]) {
				COUNTER[i] = COUNTER[i] + 1;
			}
		}
	}

	int max1Index = 300;
	int max2Index = 300;
	int max3Index = 300;
	int max4Index = 300;
	int max5Index = 300;
	int max6Index = 300;
	
	int maxIndex;
	////////////////////////////////////find top 6///////////////////////////////
	for(int i = 1; i < 7; i++) 
	{
		maxIndex = 0;
		//make sure you don't check an index you already assigned
		while(maxIndex == max1Index || maxIndex == max2Index || maxIndex == max3Index || maxIndex == max4Index || maxIndex == max5Index || maxIndex == max6Index)
		{
			maxIndex++;
		}
		for(int j = 1; j < NUM_COLORS; j++) 
		{
			//make sure you don't check an index you already assigned
			if(j == max1Index || j == max2Index || j == max3Index || j == max4Index || j == max5Index || j == max6Index) {
				;
			}
			else if(COUNTER[maxIndex] <= COUNTER[j]) {
				maxIndex = j;
			}
		}
		if(i == 1) {
			max1Index = maxIndex;
		}
		if(i == 2) {
			max2Index = maxIndex;
		}
		if(i == 3) {
			max3Index = maxIndex;
		}
		if(i == 4) {
			max4Index = maxIndex;
		}
		if(i == 5) {
			max5Index = maxIndex;
		}
		if(i == 6) {
			max6Index = maxIndex;
		}
	}

	///////////////////////////////////////////////////////
	//PRINT COLOR PALETTE
	printf("*********************************GPU***********************************\n");
	printf("Now printing the color palette of this image:\n");
	//if image has 6 colors or more:
	if(COUNTER[max1Index] != 0 && COUNTER[max2Index] != 0 && COUNTER[max3Index] != 0 
		&& COUNTER[max4Index] != 0 && COUNTER[max5Index] != 0 && COUNTER[max6Index] != 0) {
		printf("1- color#%d: %s- %d pixels\n", max1Index + 1, COLORNAMES[max1Index], COUNTER[max1Index]);
		printf("2- color#%d: %s- %d pixels\n", max2Index + 1, COLORNAMES[max2Index], COUNTER[max2Index]);
		printf("3- color#%d: %s- %d pixels\n", max3Index + 1, COLORNAMES[max3Index], COUNTER[max3Index]);
		printf("4- color#%d: %s- %d pixels\n", max4Index + 1, COLORNAMES[max4Index], COUNTER[max4Index]);
		printf("5- color#%d: %s- %d pixels\n", max5Index + 1, COLORNAMES[max5Index], COUNTER[max5Index]);
		printf("6- color#%d: %s- %d pixels\n", max6Index + 1, COLORNAMES[max6Index], COUNTER[max6Index]);
	}
	//if image has 5 colors only
	else if(COUNTER[max1Index] != 0 && COUNTER[max2Index] != 0 && COUNTER[max3Index] != 0 
		&& COUNTER[max4Index] != 0 && COUNTER[max5Index] != 0) {
		printf("1- color#%d: %s- %d pixels\n", max1Index + 1, COLORNAMES[max1Index], COUNTER[max1Index]);
		printf("2- color#%d: %s- %d pixels\n", max2Index + 1, COLORNAMES[max2Index], COUNTER[max2Index]);
		printf("3- color#%d: %s- %d pixels\n", max3Index + 1, COLORNAMES[max3Index], COUNTER[max3Index]);
		printf("4- color#%d: %s- %d pixels\n", max4Index + 1, COLORNAMES[max4Index], COUNTER[max4Index]);
		printf("5- color#%d: %s- %d pixels\n", max5Index + 1, COLORNAMES[max5Index], COUNTER[max5Index]);
		printf("image only contains 5 colors\n");
	}	
	//if image has 4 colors only
	else if(COUNTER[max1Index] != 0 && COUNTER[max2Index] != 0 && COUNTER[max3Index] != 0 
		&& COUNTER[max4Index] != 0) {
		printf("1- color#%d: %s- %d pixels\n", max1Index + 1, COLORNAMES[max1Index], COUNTER[max1Index]);
		printf("2- color#%d: %s- %d pixels\n", max2Index + 1, COLORNAMES[max2Index], COUNTER[max2Index]);
		printf("3- color#%d: %s- %d pixels\n", max3Index + 1, COLORNAMES[max3Index], COUNTER[max3Index]);
		printf("4- color#%d: %s- %d pixels\n", max4Index + 1, COLORNAMES[max4Index], COUNTER[max4Index]);
		printf("image only contains 4 colors\n");
	}
	else if(COUNTER[max1Index] != 0 && COUNTER[max2Index] != 0 && COUNTER[max3Index] != 0) { //if image has 3 colors only
		printf("1- color#%d: %s- %d pixels\n", max1Index + 1, COLORNAMES[max1Index], COUNTER[max1Index]);
		printf("2- color#%d: %s- %d pixels\n", max2Index + 1, COLORNAMES[max2Index], COUNTER[max2Index]);
		printf("3- color#%d: %s- %d pixels\n", max3Index + 1, COLORNAMES[max3Index], COUNTER[max3Index]);
		printf("image only contains 3 colors\n");
	}	
	else if(COUNTER[max1Index] != 0 && COUNTER[max2Index] != 0) { //if image has 2 colors only
		printf("1- color#%d: %s- %d pixels\n", max1Index + 1, COLORNAMES[max1Index], COUNTER[max1Index]);
		printf("2- color#%d: %s- %d pixels\n", max2Index + 1, COLORNAMES[max2Index], COUNTER[max2Index]);
		printf("image only contains 2 colors\n");
	}
	else if(COUNTER[max1Index] != 0) { //if image has 1 colors only
		printf("1- color#%d: %s- %d pixels\n", max1Index + 1, COLORNAMES[max1Index], COUNTER[max1Index]);
		printf("image only contains 1 color\n");
	}
	
	free(COUNTER);
	
	endtime("GPU");
    return 0;
}
